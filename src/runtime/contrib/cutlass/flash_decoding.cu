#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <tvm/runtime/ndarray.h>
#include <tvm/runtime/packed_func.h>
#include <tvm/runtime/registry.h>
#include <tvm/runtime/container/shape_tuple.h>

#include "../../../3rdparty/libflash_attn/include/flash.h"

namespace tvm {
namespace runtime {
namespace flash_attn {

Array<NDArray> AllocateKVCache(int head_size, int num_layers, int num_heads, int block_size,
                               int num_blocks) {
  Array<NDArray> cache;

  int device_id;
  hipGetDevice(&device_id);

  DLDevice dev{DLDeviceType::kDLCUDA, device_id};
  ShapeTuple block_shape{num_blocks, block_size, num_heads, head_size};

  for (int i = 0; i < num_layers; ++i) {
    NDArray key_blocks = NDArray::Empty(block_shape, runtime::DataType::Float(16), dev);
    NDArray value_blocks = NDArray::Empty(block_shape, runtime::DataType::Float(16), dev);
    cache.push_back(key_blocks);
    cache.push_back(value_blocks);
  }

  return cache;
}

template <typename scalar_t>
__global__ void update_cache_kernel(
    const scalar_t* __restrict__ key,          // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,        // [num_tokens, num_heads, head_size]
    scalar_t* __restrict__ key_cache,          // [num_blocks, block_size, num_heads, head_size]
    scalar_t* __restrict__ value_cache,        // [num_blocks, block_size, num_heads, head_size]
    const int* __restrict__ slot_mapping,  // [num_tokens]
    int stride, int num_heads, int head_size) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  if (slot_idx < 0) {
    // Padding token that should be ignored.
    return;
  }
  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_idx = token_idx * stride + i;
    const int64_t tgt_idx = slot_idx * n + i;
    key_cache[tgt_idx] = key[src_idx];
    value_cache[tgt_idx] = value[src_idx];
  }
}

// Grid: (num_layers, num_pairs)
template <typename scalar_t>
__global__ void copy_blocks_kernel(int64_t* key_cache_ptrs, int64_t* value_cache_ptrs,
                                   const int64_t* __restrict__ block_mapping,
                                   const int numel_per_block) {
  const int layer_idx = blockIdx.x;
  const int pair_idx = blockIdx.y;

  scalar_t* key_cache = reinterpret_cast<scalar_t*>(key_cache_ptrs[layer_idx]);
  scalar_t* value_cache = reinterpret_cast<scalar_t*>(value_cache_ptrs[layer_idx]);
  int64_t src_block_number = block_mapping[2 * pair_idx];
  int64_t dst_block_number = block_mapping[2 * pair_idx + 1];

  const int64_t src_block_offset = src_block_number * numel_per_block;
  const int64_t dst_block_offset = dst_block_number * numel_per_block;
  for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
    int64_t src_offset = src_block_offset + i;
    int64_t dst_offset = dst_block_offset + i;
    key_cache[dst_offset] = key_cache[src_offset];
  }
  for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
    int64_t src_offset = src_block_offset + i;
    int64_t dst_offset = dst_block_offset + i;
    value_cache[dst_offset] = value_cache[src_offset];
  }
}

template <typename scalar_t>
__global__ void reconstruct_from_cache_kernel(
    const scalar_t* __restrict__ key_cache,  // [num_blocks, block_size, num_heads, head_size]
    const scalar_t* __restrict__ value_cache,  // [num_blocks, block_size, num_heads, head_size]
    const int* __restrict__ slot_mapping,      // [num_tokens]
    scalar_t* __restrict__ key,                // [num_tokens, num_heads, head_size]
    scalar_t* __restrict__ value,              // [num_tokens, num_heads, head_size]
    const int stride, const int num_heads, const int head_size, const int block_size) {
  const int token_idx = blockIdx.x;
  const int slot_idx = slot_mapping[token_idx];

  if (slot_idx < 0) {
    // Padding token that should be ignored.
    return;
  }

  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t tgt_idx = token_idx * stride + i;
    const int64_t src_idx = slot_idx * n + i;
    key[tgt_idx] = key_cache[src_idx];
    value[tgt_idx] = value_cache[src_idx];
  }

}

}  // namespace flash_attn

/*
  query: (batch_size, seqlen_q, num_heads, head_size), fp16
  key_cache: (num_blocks, page_block_size, num_heads_k, head_size), fp16
  value_cache: num_blocks, page_block_size, num_heads_k, head_size), fp16
  block_tables: (batch_size, max_num_blocks_per_seq), int32
  context_lens: (batch_size,), int32
  softmax_lse_accum: (max_num_splits, batch_size, num_heads, seqlen_q), fp32
  output_accum: (max_num_splits, batch_size, num_heads, seqlen_q, head_size), fp32
  out: (batch_size, seqlen_q, num_heads, head_size), fp16
*/
TVM_REGISTER_GLOBAL("tvm.contrib.flash_attn.flash_decoding_with_paged_kvcache")
  .set_body_typed([](const DLTensor* query, const DLTensor* key_cache,
                     const DLTensor* value_cache, const DLTensor* block_tables,
                     const DLTensor* context_lens, DLTensor* softmax_lse_accum,
                     DLTensor* output_accum, DLTensor* out) {
      int batch_size = query->shape[0];
      int seqlen_q = query->shape[1];
      int num_heads = query->shape[2];
      int head_dim = query->shape[3];
      int num_heads_k = key_cache->shape[2];
      int num_blocks = key_cache->shape[0];
      int block_size = key_cache->shape[1];
      int max_num_blocks_per_seq = block_tables->shape[1];
      float softmax_scale = 1.0 / sqrt(static_cast<float>(head_dim));

      ICHECK(block_size % 64 == 0) << "Block size needs to be a multiple of 64.";

      auto block_table_ptr = static_cast<int*>(block_tables->data);
      auto seqlens_k_ptr = static_cast<int*>(context_lens->data);

      using half = ::flash_attn::half;

      ICHECK(TypeMatch(block_tables->dtype, kDLInt, 32));
      ICHECK(TypeMatch(context_lens->dtype, kDLInt, 32));
      ICHECK(TypeMatch(softmax_lse_accum->dtype, kDLFloat, 32));
      ICHECK(TypeMatch(output_accum->dtype, kDLFloat, 32));

      auto q_ptr = static_cast<half*>(query->data);
      auto kcache_ptr = static_cast<half*>(key_cache->data);
      auto vcache_ptr = static_cast<half*>(value_cache->data);
      auto softmax_lse_accum_ptr = static_cast<float*>(softmax_lse_accum->data);
      auto output_accum_ptr = static_cast<float*>(output_accum->data);
      auto output_ptr = static_cast<half*>(out->data);

      int q_head_stride = head_dim;
      int k_head_stride = head_dim;
      int v_head_stride = head_dim;
      int o_head_stride = head_dim;
      int q_row_stride = q_head_stride * num_heads;
      int k_row_stride = k_head_stride * num_heads_k;
      int v_row_stride = v_head_stride * num_heads_k;
      int o_row_stride = o_head_stride * num_heads;
      int q_batch_stride = q_row_stride * seqlen_q;
      int k_batch_stride = k_row_stride * block_size;
      int v_batch_stride = v_row_stride * block_size;
      int o_batch_stride = o_row_stride * seqlen_q;
      int block_table_batch_stride = max_num_blocks_per_seq;

      ::flash_attn::flash_attention_splitkv_paged_forward(
          q_ptr, kcache_ptr, vcache_ptr, block_table_ptr, seqlens_k_ptr,
          softmax_lse_accum_ptr, output_accum_ptr,
          output_ptr, batch_size, seqlen_q, num_heads, num_heads_k, head_dim,
          q_batch_stride,
          k_batch_stride,
          v_batch_stride,
          o_batch_stride,
          q_head_stride,
          k_head_stride,
          v_head_stride,
          o_head_stride,
          q_row_stride,
          k_row_stride,
          v_row_stride,
          o_row_stride,
          num_blocks, block_size, max_num_blocks_per_seq,
          block_table_batch_stride,
          softmax_scale,
          true /* is_causal*/);
    });

TVM_REGISTER_GLOBAL("tvm.contrib.flash_attn.allocate_kv_cache").set_body_typed(flash_attn::AllocateKVCache);

TVM_REGISTER_GLOBAL("tvm.contrib.flash_attn.update_cache")
    .set_body_typed([](NDArray key, NDArray value, NDArray key_cache, NDArray value_cache,
                       NDArray slot_mapping) {
      int num_tokens = key->shape[0];
      int num_heads = key->shape[1];
      int head_size = key->shape[2];
      int stride = key->shape[1] * key->shape[2];

      dim3 grid(num_tokens);
      dim3 block(std::min(num_heads * head_size, 512));

      using scalar_t = uint16_t;

      flash_attn::update_cache_kernel<scalar_t><<<grid, block>>>(
          static_cast<const scalar_t*>(key->data),
	  static_cast<const scalar_t*>(value->data),
          static_cast<scalar_t*>(key_cache->data),
	  static_cast<scalar_t*>(value_cache->data),
          static_cast<const int*>(slot_mapping->data),
	  stride, num_heads, head_size);

      return Array{key_cache, value_cache};
    });

TVM_REGISTER_GLOBAL("tvm.contrib.flash_attn.copy_blocks")
    .set_body_typed([](Array<NDArray> key_value_caches, NDArray block_mapping) {
      auto num_layers = key_value_caches.size() / 2;
      auto num_pairs = block_mapping->shape[0] / 2;

      if (num_layers == 0) {
        return;
      }

      std::vector<int64_t> key_cache_ptrs(num_layers);
      std::vector<int64_t> value_cache_ptrs(num_layers);
      for (size_t layer_idx = 0; layer_idx < num_layers; ++layer_idx) {
        key_cache_ptrs[layer_idx] =
            reinterpret_cast<int64_t>(key_value_caches[2 * layer_idx]->data);
        value_cache_ptrs[layer_idx] =
            reinterpret_cast<int64_t>(key_value_caches[2 * layer_idx + 1]->data);
      }

      NDArray key_cache = key_value_caches[1];  // [num_blocks, num_heads, head_size, block_size]
      DLDevice dev = key_cache->device;

      NDArray key_cache_ptrs_gpu =
          NDArray::Empty({static_cast<int>(num_layers)}, runtime::DataType::Int(64), dev);
      NDArray value_cache_ptrs_gpu =
          NDArray::Empty({static_cast<int>(num_layers)}, runtime::DataType::Int(64), dev);
      key_cache_ptrs_gpu.CopyFromBytes(key_cache_ptrs.data(),
                                       sizeof(int64_t) * key_cache_ptrs.size());
      value_cache_ptrs_gpu.CopyFromBytes(value_cache_ptrs.data(),
                                         sizeof(int64_t) * value_cache_ptrs.size());

      NDArray block_mapping_gpu =
          NDArray::Empty(block_mapping.Shape(), runtime::DataType::Int(64), dev);
      block_mapping_gpu.CopyFromBytes(block_mapping->data,
                                      sizeof(int64_t) * block_mapping->shape[0]);

      const int numel_per_block = key_cache->shape[1] * key_cache->shape[2] * key_cache->shape[3];
      dim3 grid(num_layers, num_pairs); dim3 block(std::min(1024, numel_per_block));

      using scalar_t = uint16_t;
      flash_attn::copy_blocks_kernel<scalar_t>
          <<<grid, block>>>(static_cast<int64_t*>(key_cache_ptrs_gpu->data),
                            static_cast<int64_t*>(value_cache_ptrs_gpu->data),
                            static_cast<int64_t*>(block_mapping_gpu->data), numel_per_block);
    });

TVM_REGISTER_GLOBAL("tvm.contrib.flash_attn.reconstruct_from_cache")
    .set_body_typed([](NDArray key_cache, NDArray value_cache, NDArray slot_mapping, NDArray key, NDArray value) {
      int num_tokens = slot_mapping->shape[0];
      int num_heads = value_cache->shape[2];
      int head_size = value_cache->shape[3];
      int block_size = value_cache->shape[1];

      CHECK_EQ(num_tokens, value->shape[0]);
      CHECK_EQ(num_heads, value->shape[1]);
      CHECK_EQ(head_size, value->shape[2]);
      CHECK_EQ(key->shape[0], value->shape[0]);
      CHECK_EQ(key->shape[1], value->shape[1]);
      CHECK_EQ(key->shape[2], value->shape[2]);

      int stride = key->shape[1] * key->shape[2];

      dim3 grid(num_tokens);
      dim3 block(std::min(num_heads * head_size, 512));

      using scalar_t = uint16_t;
      flash_attn::reconstruct_from_cache_kernel<scalar_t>
          <<<grid, block>>>(static_cast<const scalar_t*>(key_cache->data),
                            static_cast<const scalar_t*>(value_cache->data),
                            static_cast<const int*>(slot_mapping->data),
                            static_cast<scalar_t*>(key->data), static_cast<scalar_t*>(value->data),
                            stride, num_heads, head_size, block_size);

    });

}  // namespace runtime
}  // namespace tvm
